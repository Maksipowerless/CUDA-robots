#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <cmath>
#include <vector>
#include <stdlib.h>
#include"/opt/software/nvidia/cuda-8.0/include/hip/hip_runtime.h"
#include <stdio.h>

using namespace std;
#define N 100
#define threads 16

__device__ int MAP[N][N];

__global__ void fillMap(int map[N][N])
{
	for(int i=0; i<N; i++)
	{
		for(int j=0; j<N; j++)
		{
		   MAP[i][j] = map[i][j];
		}
	}
}

__device__ void bresenhamLine(int x1, int y1, int x2, int y2)
{
    const int heightOfRobot = 50;
    const int heightOfStantion = 120;
    const int deltaX = abs(x2 - x1);
    const int deltaY = abs(y2 - y1);
    const int signX = x1 < x2 ? 1 : -1;
    const int signY = y1 < y2 ? 1 : -1;
    int error = deltaX - deltaY;
    int count = 0;//cчеткик кол-ва точек

    int numOfVersities[1000];//массив с номерами точек
    int heightOfVersities[1000];//массив с высотами точек
    int xCoord[1000];//массив с x координатами
    int yCoord[1000];//массив с y координатами

    for(int i=0; i< 1000; i++)
    {
        numOfVersities[i]=-999;
        heightOfVersities[i]=-999;
        xCoord[i]=-999;
        yCoord[i]=-999;
    }

    while(x1 != x2 || y1 != y2)
    {
        count++;
        const int error2 = error * 2;
        if(error2 > -deltaY)
        {
            error -= deltaY;
            x1 += signX;
        }
        if(error2 < deltaX)

        {
            error += deltaX;
            y1 += signY;
        }

        numOfVersities[count-1]=count;
        heightOfVersities[count-1]=MAP[x1][y1];
        xCoord[count-1]=x1;
        yCoord[count-1]=y1;
    }

    float alpha = 0;
    int tempNumber = 0;
    int tempHeight = 0;
    int tempX = 0;
    int tempY = 0;

    while(numOfVersities[0] != -999)
    {
        int number  = 0;//номер вершины в массивеm, через которую удаляют невидимые вершины
        //поиск точки с меньшим полярным углом от стационарной
        for(int i=0; i<count-1; i++)
        {
            float A = numOfVersities[i];
            float B = sqrtf(numOfVersities[i]*numOfVersities[i] + 
                          (heightOfStantion - heightOfVersities[i])*(heightOfStantion - heightOfVersities[i]));
            float temp = 0;
            temp = A/B;
            if(temp > alpha)
            {
                alpha = temp;
                tempNumber = numOfVersities[i];
                tempHeight = heightOfVersities[i];
                tempX = xCoord[i];
                tempY = yCoord[i];
                number = i;
            }
        }

        //удаление невидимых точек за текущей
        for(int i = number + 1; numOfVersities[i] != -999; i++)
        {
            if(heightOfVersities[i] + heightOfRobot < tempHeight)
            {
                MAP[xCoord[i]][yCoord[i]] = 999;

                numOfVersities[i]=-999;
                heightOfVersities[i]=-999;
                xCoord[i]=-999;
                yCoord[i]=-999;

                for(int j=i; j<count-1; j++)
                {
                    int temp =  numOfVersities[j];
                    numOfVersities[j] = numOfVersities[j+1];
                    numOfVersities[j+1] = temp;

                    temp =  heightOfVersities[j];
                    heightOfVersities[j] = heightOfVersities[j+1];
                    heightOfVersities[j+1] = temp;

                    temp =  xCoord[j];
                    xCoord[j] = xCoord[j+1];
                    xCoord[j+1] = temp;

                    temp =  yCoord[j];
                    yCoord[j] = yCoord[j+1];
                    yCoord[j+1] = temp;

                }
                count--;
            }
        }

        //удаление текущей точки
        numOfVersities[number]=-999;
        heightOfVersities[number]=-999;
        xCoord[number]=-999;
        yCoord[number]=-999;

        for(int j=number; j<count-1; j++)
        {
            int temp =  numOfVersities[j];
            numOfVersities[j] = numOfVersities[j+1];
            numOfVersities[j+1] = temp;

            temp =  heightOfVersities[j];
            heightOfVersities[j] = heightOfVersities[j+1];
            heightOfVersities[j+1] = temp;

            temp =  xCoord[j];
            xCoord[j] = xCoord[j+1];
            xCoord[j+1] = temp;

            temp =  yCoord[j];
            yCoord[j] = yCoord[j+1];
            yCoord[j+1] = temp;

        }

        alpha = 0;
    }
}

__global__ void findLine()
{

 int xStatic = 66;
 int yStatic = 55;
 int tid = blockIdx.x;
	
 bresenhamLine(xStatic, yStatic, tid,0);
 bresenhamLine(xStatic, yStatic, tid, N-1);
 bresenhamLine(xStatic, yStatic, 0, tid);
 bresenhamLine(xStatic, yStatic, N-1, tid);
 

	
       
}


int map[N][N];
 
int main()
{
 int heightOfStantion = 120;
 int xStatic = 66;
 int yStatic = 55;

    srand(time(0));
       for(int i=0; i<N; i++)
        for(int j=0; j<N; j++)
            map[i][j] = rand() % 80 +5;
   
  void* a_DATA;
  hipGetSymbolAddress(&a_DATA, HIP_SYMBOL(MAP));
  hipMemcpy(a_DATA, map, sizeof(map), hipMemcpyHostToDevice);
  findLine <<<N,N>>>();
  hipMemcpy(map, a_DATA, sizeof(map), hipMemcpyDeviceToHost);   

	
for(int i=0; i< N; i++)
	{
		for(int j=0; j< N; j++)
		{
			printf("%d",map[i][j]);
			printf(" ");
		}
		printf("\n");
	}

    return 0;
}
