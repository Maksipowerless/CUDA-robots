#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <cmath>
#include <vector>
#include<stdlib.h>
#include"/opt/software/nvidia/cuda-8.0/include/hip/hip_runtime.h"

using namespace std;
#define N 1000
#define threads 10

int map[N][N];
int heightOfRobot = 50;
int heightOfStantion = 120;
int xStatic = 66;
int yStatic = 55;


--global__ void findLine(int** a)
{
 int tid = blockIdx.x;
	
    for(int i=0; i<N; i++)
	{
	 if(tid < threads)
	 {
	   bresenhamLine(xStatic, yStatic, tid,0);
	   bresenhamLine(xStatic, yStatic, tid, N-1);
           bresenhamLine(xStatic, yStatic, 0, tid);
           bresenhamLine(xStatic, yStatic, N-1, tid);
	 }
       }
}

void findDarkArea(vector<pair<int, int> >& coord, vector<pair<int, int> >& versities)
{
    float alpha = 0;
    pair<int,int> tempPair;
    pair<int,int> currentPoint;

    while(versities.size() != 0)
    {
        //поиск точки с меньшим полярным углом от стационарной
        for(int i=0; i<versities.size(); i++)
        {
            float A = versities[i].first;
            float B = sqrt(pow(versities[i].first, 2) + pow((heightOfStantion - versities[i].second), 2));
            float temp = 0;
            temp = A/B;
            if(temp > alpha)
            {
                alpha = temp;
                tempPair = coord[i];
                currentPoint = versities[i];
            }
        }

        //проверка на видимость всех точек за текущей
	int number  = 0;
	for(int j=0; j<versities.size(); j++)
	{
	  if(versities[j] == currentPoint)
		{
		  number = j;
		  break;
		}
	}	

        for(int i = number + 1; i< versities.size(); i++)
        {
            if(versities[i].second + heightOfRobot < currentPoint.second)
            {
                map[coord[i].first][coord[i].second] = 999;
                coord.erase(coord.begin() + i);
                versities.erase(versities.begin() + i);
            }
        }

	for(int j=0; j<coord.size(); j++)
	{
		if(coord[j] == tempPair)
		{
		  coord.erase(coord.begin() + j);
		  versities.erase(versities.begin() +j);
		  break;
		}
	}
    
        alpha = 0;
    }
}


void bresenhamLine(int x1, int y1, int x2, int y2)
{
    const int deltaX = abs(x2 - x1);
    const int deltaY = abs(y2 - y1);
    const int signX = x1 < x2 ? 1 : -1;
    const int signY = y1 < y2 ? 1 : -1;
    int error = deltaX - deltaY;
    int count = 0;//cчеткик кол-ва точек

    vector<pair<int,int> > versities;
    vector<pair<int,int> > coord;

    while(x1 != x2 || y1 != y2)
    {
        count++;
        const int error2 = error * 2;
        if(error2 > -deltaY)
        {
            error -= deltaY;
            x1 += signX;
        }
        if(error2 < deltaX)

        {
            error += deltaX;
            y1 += signY;
        }

        pair<int,int> pair;
        pair.first = count;
        pair.second = map[x1][y1];
        versities.push_back(pair);

        pair.first = x1;
        pair.second = y1;
        coord.push_back(pair);
    }
	findDarkArea(coord, versities);
}

int main()
{
    srand(time(0));
       for(int i=0; i<N; i++)
        for(int j=0; j<N; j++)
            map[i][j] = rand() % 80 +5;
   
    map[xStatic][yStatic] = heightOfStantion;

	int** dev_a;
	hipMalloc((void***) &dev_a, threads*sizeof(int));
	for(int i=0; i<threads; i++)
	{
		hipMalloc((void**) &dev_a[i], threads*sizeof(int)); 
	}
	
	for(int i=0; i< N; i++)
	{
		for(int j=0; j< N; j++)
		{
			cout<<map[i][j] <<" ";
		}
		cout<<"\n";
	}
    return 0;
}
