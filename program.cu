#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <cmath>
#include <vector>
#include <stdlib.h>
#include"/opt/software/nvidia/cuda-8.0/include/hip/hip_runtime.h"
#include <stdio.h>

using namespace std;
#define N 1000
#define threads 74  
#define blocks 54
#define X 500
#define Y 500git 

__device__ int MAP[N][N];//карта на GPU
__device__ int SUM = 4*N-4;//количество крайних точек

__device__ void bresenhamLine(int x1, int y1, int x2, int y2)
{
    const int heightOfRobot = 50;
    const int heightOfStantion = 120;
    const int deltaX = abs(x2 - x1);
    const int deltaY = abs(y2 - y1);
    const int signX = x1 < x2 ? 1 : -1;
    const int signY = y1 < y2 ? 1 : -1;
    int error = deltaX - deltaY;
    int count = 0;//cчеткик кол-ва точек

    int numOfVersities[1000];//массив с номерами точек
    int heightOfVersities[1000];//массив с высотами точек
    int xCoord[1000];//массив с x координатами
    int yCoord[1000];//массив с y координатами
    
      while(x1 != x2 || y1 != y2)
    {
        count++;
        const int error2 = error * 2;
        if(error2 > -deltaY)
        {
            error -= deltaY;
            x1 += signX;
        }
        if(error2 < deltaX)

        {
            error += deltaX;
            y1 += signY;
        }

        numOfVersities[count-1]=count;
        heightOfVersities[count-1]=MAP[x1][y1];
        xCoord[count-1]=x1;
        yCoord[count-1]=y1;
    }

    float alpha = 0;
    int tempNumber = 0;
    int tempHeight = 0;
    int tempX = 0;
    int tempY = 0;
    int sizeOfLine = count;//кол-во точек в прямой
    count = 0;//количество удаленных точек
    int delElement = -99;//признак удаленного элемента в массиве

    while(count != sizeOfLine)
    {
        int number  = 0;//номер вершины в массиве, через которую удаляют невидимые вершины
        //поиск точки с меньшим полярным углом от стационарной
        for(int i=0; i < sizeOfLine; i++)
        {
            if(numOfVersities[i] != delElement)
            {
                float A = numOfVersities[i];
                float B = sqrtf(numOfVersities[i]*numOfVersities[i] + 
                          (heightOfStantion - heightOfVersities[i])*(heightOfStantion - heightOfVersities[i]));
                float temp = A/B;
                if(temp > alpha)
                {
                    alpha = temp;
                    tempNumber = numOfVersities[i];
                    tempHeight = heightOfVersities[i];
                    tempX = xCoord[i];
                    tempY = yCoord[i];
                    number = i;
                }
            }
        }

        //удаление невидимых точек за текущей
        for(int i = number + 1; i < sizeOfLine; i++)
        {
            if(heightOfVersities[i] != delElement && heightOfVersities[i] + heightOfRobot < tempHeight)
            {
                MAP[xCoord[i]][yCoord[i]] = -999;
                numOfVersities[i]= delElement;
                heightOfVersities[i]= delElement;
                count++;
            }
        }

        //удаление текущей точки
        numOfVersities[number]= delElement;
        heightOfVersities[number]= delElement;
        count++;

        alpha = 0;
    }
}

__global__ void findLine()
{

 int xStatic = X;
 int yStatic = Y;
 int tid = threadIdx.x;
 int bid = blockIdx.x;
for(int i = (bid*threads+tid)*SUM/(threads*blocks); i < (bid*threads+tid+1)*SUM/(threads*blocks); i++)
{ 
	int column = 0;
	int str = 0;

	if(i <= SUM/4)
	{
	  column = i;
	}
	else if(i <=  SUM/2)
	{
	  column = N-1;
	  str = i-(N-1);
	}
	else if(i <= 3*SUM/4)
	{
	  str = N-1;
	  column = N-1-i+2*(N-1);
	}
	else
	{
	   str = i-3*(N-1);
	}
	

      bresenhamLine(xStatic, yStatic, str, column);
     //MAP[str][column]=0;
}
 //  __syncthreads();
}


int map[N][N];
 
int main()
{
    srand(time(0));
       for(int i=0; i<N; i++)
        for(int j=0; j<N; j++)
            map[i][j] = rand() % 80 +5;
   
/*
for(int i=0; i< N; i++)
	{
		for(int j=0; j< N; j++)
		{
			printf("%d",map[i][j]);
			printf(" ");
		}
		printf("\n");
	}
  printf("\n");
*/
  hipSetDevice(0);

 hipEvent_t timStart, timCopyTo, timStopWork, timCopyFrom;
 hipEventCreate(&timStart);
 hipEventCreate(&timCopyTo);
 hipEventCreate(&timStopWork);
 hipEventCreate(&timCopyFrom);

 hipEventRecord(timStart);
 	
  void* a_DATA;
  hipGetSymbolAddress(&a_DATA, HIP_SYMBOL(MAP));
  hipMemcpy(a_DATA, map, sizeof(map), hipMemcpyHostToDevice);

  hipEventRecord(timCopyTo);

  dim3 numThreads = dim3(threads);
  dim3 numBlocks = dim3(blocks);

  findLine<<<numBlocks, numThreads>>>();

  hipEventRecord(timStopWork);

  hipMemcpy(map, a_DATA, sizeof(map), hipMemcpyDeviceToHost);   

  hipEventRecord(timCopyFrom);
/*
for(int i=0; i< N; i++)
	{
		for(int j=0; j< N; j++)
		{
			printf("%d",map[i][j]);
			printf(" ");
		}
		printf("\n");
	}
*/
float t1,t2,t3;
hipEventElapsedTime(&t1,timStart, timCopyTo);
hipEventElapsedTime(&t2, timCopyTo,timStopWork);
hipEventElapsedTime(&t3,timStopWork, timCopyFrom);

cout<< "\n"<<t1 << " "<<t2<<" "<< t3 << "\n";
    return 0;
}
